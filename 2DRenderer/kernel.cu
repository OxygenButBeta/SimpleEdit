#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void grayscaleKernel(unsigned char* rgbData, unsigned char* grayData, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 3;  // RGB için 3 kanal
        unsigned char r = rgbData[idx];
        unsigned char g = rgbData[idx + 1];
        unsigned char b = rgbData[idx + 2];

        // Gri tonlama formülü: (R + G + B) / 3
        grayData[y * width + x] = (r + g + b) / 3;
    }
}

extern "C" void runGrayscale(unsigned char* rgbData, unsigned char* grayData, int width, int height) {
    unsigned char* d_rgbData;
    unsigned char* d_grayData;

    size_t rgbSize = width * height * 3 * sizeof(unsigned char);
    size_t graySize = width * height * sizeof(unsigned char);

    // GPU'ya bellek tahsisi
    hipMalloc(&d_rgbData, rgbSize);
    hipMalloc(&d_grayData, graySize);

    // CPU'dan GPU'ya veri kopyalama
    hipMemcpy(d_rgbData, rgbData, rgbSize, hipMemcpyHostToDevice);

    // Kernel'i çalıştırma (16x16 blok boyutu kullandık)
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    grayscaleKernel <<<gridDim, blockDim >> > (d_rgbData, d_grayData, width, height);

    // Sonuçları GPU'dan CPU'ya geri kopyalama
    hipMemcpy(grayData, d_grayData, graySize, hipMemcpyDeviceToHost);

    // Belleği serbest bırakma
    hipFree(d_rgbData);
    hipFree(d_grayData);
}
int main()
{
	return 0;
}